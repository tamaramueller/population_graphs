#include "hip/hip_runtime.h"
// test convolution with autodiff
// compile with
//		nvcc -I.. -DCUDA_BLOCK_SIZE=192 -DMAXTHREADSPERBLOCK0=1024 -DSHAREDMEMPERBLOCK0=49152 -Wno-deprecated-gpu-targets -std=c++14 -O2 -o build/test_autodiff test_autodiff.cu

// we define an arbitrary function using available blocks,
// then test its convolution on the CPU or GPU, then get its gradients and test again the convolutions

// Here we build the function f(x,y,u,v,beta) = <u,v>^2 * exp(-p*|x-y|^2) * beta
// where p is a scalar parameter, x, y, beta are 3D vectors, and u, v are 4D vectors
// and the convolution is res_i = sum_j f(x_i,y_j,u_i,v_j,beta_j)
// then we define the gradients of this reduction with respect to x and y 
// (i.e. the gradient of x -> sum_j f(x_i,y_j,...) and y -> sum_j f(x_i,y_j,...)), with new input variable eta (3D).

#include <algorithm>

// __TYPE__ is floating point type used in all code; may be float or double
// if not defined here or previously, it will be set by default to float in keops_includes.h below
#ifndef __TYPE__
  #define __TYPE__ double
#endif

#include <keops_includes.h>

using namespace keops;

__TYPE__ floatrand() {
    return ((__TYPE__) std::rand())/RAND_MAX-.5;    // random value between -.5 and .5
}

template < class V > void fillrandom(V& v) {
    generate(v.begin(), v.end(), floatrand);    // fills vector with random values
}



int main() {

    // set Gpu device
    int deviceID = 0;
    
    // In this part we define the symbolic variables of the function
    auto p = Pm(0,1);	 // p is the first variable and is a scalar parameter
    auto x = Vi(1,3); 	 // x is the second variable and represents a 3D vector, "i"-indexed.
    auto y = Vj(2,3); 	 // y is the third variable and represents a 3D vector, "j"-indexed.
    auto u = Vi(3,4); 	 // u is the fourth variable and represents a 4D vector, "i"-indexed.
    auto v = Vj(4,4); 	 // v is the fourth variable and represents a 4D vector, "j"-indexed.
    auto beta = Vj(5,3); // beta is the sixth variable and represents a 3D vector, "j"-indexed.

    // symbolic expression of the function ------------------------------------------------------

    // here we define f = <u,v>^2 * exp(-p*|x-y|^2) * beta in usual notations
    auto f = Square(u|v) * Exp(-p*SqNorm2(x-y)) * beta;
    
    // We define the reduction operation on f. Here a sum reduction, performed over the "j" index, and resulting in a "i"-indexed variable
    auto Sum_f = Sum_Reduction(f,0);  // 0 means output of reduction will be "i"-indexed (0 means"i", 1 means "j")

    // Now we define gradients of the reduction operation:
    // First we define a new variable to be the input of gradient
    auto eta = Vi(6,Sum_f.DIM); 
    // now we gradient with respect to x ---------------------------------------------------------------
    auto Grad_x_Sum_f = Grad(Sum_f,x,eta);
    // and gradient with respect to y  --------------------------------------------------------------
    auto Grad_y_Sum_f = Grad(Sum_f,y,eta);



    // now we test ------------------------------------------------------------------------------

    int Nx=5000, Ny=2000;

    // here we define actual data for all variables and feed it it with random values
    std::vector<__TYPE__> vx(Nx*x.DIM);    fillrandom(vx); __TYPE__ *px = vx.data();
    std::vector<__TYPE__> vy(Ny*y.DIM);    fillrandom(vy); __TYPE__ *py = vy.data();
    std::vector<__TYPE__> vu(Nx*u.DIM);    fillrandom(vu); __TYPE__ *pu = vu.data();
    std::vector<__TYPE__> vv(Ny*v.DIM);    fillrandom(vv); __TYPE__ *pv = vv.data();
    std::vector<__TYPE__> vb(Ny*beta.DIM); fillrandom(vb); __TYPE__ *pb = vb.data();

    // also a vector for the output
    std::vector<__TYPE__> vres(Nx*Sum_f.DIM);    fillrandom(vres); __TYPE__ *pres = vres.data();

    // and three vectors to get copies of the output, for comparing Cpu vs Gpu/1D vs Gpu/2D computations
    std::vector<__TYPE__> resgpu2D(Nx*Sum_f.DIM), resgpu1D(Nx*Sum_f.DIM), rescpu(Nx*Sum_f.DIM);

    // parameter variable
    __TYPE__ params[1];
    __TYPE__ Sigma = 4.0;
    params[0] = 1.0/(Sigma*Sigma);

    clock_t begin, end;

    begin = clock();
    hipSetDevice(deviceID);        
    end = clock();
    std::cout << "time for GPU initialization : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    std::cout << "blank run" << std::endl;
    begin = clock();
    EvalRed<GpuConv2D_FromHost>(Sum_f,Nx, Ny, pres, params, px, py, pu, pv, pb);
    end = clock();
    std::cout << "time for blank run : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    std::cout << "testing reduction" << std::endl;
    begin = clock();
    EvalRed<GpuConv2D_FromHost>(Sum_f,Nx, Ny, pres, params, px, py, pu, pv, pb);
    end = clock();
    std::cout << "time for GPU computation (2D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu2D = vres;

    begin = clock();
    EvalRed<GpuConv1D_FromHost>(Sum_f,Nx, Ny, pres, params, px, py, pu, pv, pb);
    end = clock();
    std::cout << "time for GPU computation (1D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu1D = vres;

    begin = clock();
    EvalRed<CpuConv>(Sum_f,Nx, Ny, pres, params, px, py, pu, pv, pb);
    end = clock();
    std::cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    rescpu = vres;

    // display mean of errors
    __TYPE__ s = 0;
    for(int i=0; i<Nx*Sum_f.DIM; i++)
        s += std::abs(resgpu2D[i]-rescpu[i]);
    std::cout << "mean abs error 2D =" << s/Nx << std::endl;

    s = 0;
    for(int i=0; i<Nx*Sum_f.DIM; i++)
        s += std::abs(resgpu1D[i]-rescpu[i]);
    std::cout << "mean abs error 1D =" << s/Nx << std::endl;



    rescpu.resize(Nx*Grad_x_Sum_f.DIM);
    resgpu2D.resize(Nx*Grad_x_Sum_f.DIM);
    resgpu1D.resize(Nx*Grad_x_Sum_f.DIM);
    vres.resize(Nx*Grad_x_Sum_f.DIM);
    pres = vres.data();

    std::vector<__TYPE__> ve(Nx*eta.DIM); fillrandom(ve); __TYPE__ *pe = ve.data();

    std::cout << "testing gradient wrt x" << std::endl;
    begin = clock();
    EvalRed<GpuConv2D_FromHost>(Grad_x_Sum_f,Nx, Ny,pres, params, px, py, pu, pv, pb, pe);
    end = clock();
    std::cout << "time for GPU computation (2D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu2D = vres;

    begin = clock();
    EvalRed<GpuConv1D_FromHost>(Grad_x_Sum_f,Nx, Ny, pres, params, px, py, pu, pv, pb, pe);
    end = clock();
    std::cout << "time for GPU computation (1D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu1D = vres;

    begin = clock();
    EvalRed<CpuConv>(Grad_x_Sum_f,Nx, Ny, pres, params, px, py, pu, pv, pb, pe);
    end = clock();
    std::cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    rescpu = vres;

    // display mean of errors
    s = 0;
    for(int i=0; i<Nx*Grad_x_Sum_f.DIM; i++)
        s += std::abs(resgpu2D[i]-rescpu[i]);
    std::cout << "mean abs error 2D =" << s/Nx << std::endl;

    s = 0;
    for(int i=0; i<Nx*Grad_x_Sum_f.DIM; i++)
        s += std::abs(resgpu1D[i]-rescpu[i]);
    std::cout << "mean abs error 1D =" << s/Nx << std::endl;



    // gradient wrt y, which is a "j" variable.

    rescpu.resize(Ny*Grad_y_Sum_f.DIM);
    resgpu2D.resize(Ny*Grad_y_Sum_f.DIM);
    resgpu1D.resize(Ny*Grad_y_Sum_f.DIM);
    vres.resize(Ny*Grad_y_Sum_f.DIM);
    pres = vres.data();

    std::cout << "testing gradient wrt y" << std::endl;
    begin = clock();
    EvalRed<GpuConv2D_FromHost>(Grad_y_Sum_f,Ny, Nx, pres, params, px, py, pu, pv, pb, pe);
    end = clock();
    std::cout << "time for GPU computation (2D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu2D = vres;

    begin = clock();
    EvalRed<GpuConv1D_FromHost>(Grad_y_Sum_f,Ny, Nx, pres, params, px, py, pu, pv, pb, pe);
    end = clock();
    std::cout << "time for GPU computation (1D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu1D = vres;

    begin = clock();
    EvalRed<CpuConv>(Grad_y_Sum_f,Ny, Nx, pres, params, px, py, pu, pv, pb, pe);
    end = clock();
    std::cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    rescpu = vres;

    // display mean of errors
    s = 0;
    for(int i=0; i<Ny*Grad_y_Sum_f.DIM; i++)
        s += std::abs(resgpu2D[i]-rescpu[i]);
    std::cout << "mean abs error 2D=" << s/Ny << std::endl;

    s = 0;
    for(int i=0; i<Ny*Grad_y_Sum_f.DIM; i++)
        s += std::abs(resgpu1D[i]-rescpu[i]);
    std::cout << "mean abs error 1D=" << s/Ny << std::endl;



}



