
#include <hip/hip_runtime.h>
#include <cstdio>
int main()
{
  int count = 0;
  if (hipSuccess != hipGetDeviceCount(&count)) return -1;
  if (count == 0) return -1;
  std::printf("-DMAXIDGPU=%d;",count-1);
  for (int device = 0; device < count; ++device)
  {
    hipDeviceProp_t prop;
    if (hipSuccess == hipGetDeviceProperties(&prop, device))
      std::printf("-DMAXTHREADSPERBLOCK%d=%d;-DSHAREDMEMPERBLOCK%d=%d;", device, (int)prop.maxThreadsPerBlock, device, (int)prop.sharedMemPerBlock);
  }
  return 0;
}
